/**
 * @copyright
 * Copyright      2022  Xiaomi Corporation (authors: Wei Kang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "fast_rnnt/python/csrc/fast_rnnt.h"
#include "fast_rnnt/python/csrc/mutual_information.h"

PYBIND11_MODULE(_fast_rnnt, m) {
  m.doc() = "Python wrapper for Fast Rnnt.";

  fast_rnnt::PybindMutualInformation(m);
}
